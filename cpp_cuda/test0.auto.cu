#include "hip/hip_runtime.h"
using default_int = int;
using default_uint = unsigned int;
template <typename el>
struct sptr // Shared pointer for the Spiral datatypes. They have to have the refc field inside them to work.
{
    el* base;

    __host__ __device__ sptr() : base(nullptr) {}
    __host__ __device__ sptr(el* ptr) : base(ptr) { this->base->refc++; }

    __host__ __device__ ~sptr()
    {
        if (this->base != nullptr && --this->base->refc == 0)
        {
            delete this->base;
            this->base = nullptr;
        }
    }

    __host__ __device__ sptr(sptr& x)
    {
        this->base = x.base;
        this->base->refc++;
    }

    __host__ __device__ sptr(sptr&& x)
    {
        this->base = x.base;
        x.base = nullptr;
    }

    __host__ __device__ sptr& operator=(sptr& x)
    {
        if (this->base != x.base)
        {
            delete this->base;
            this->base = x.base;
            this->base->refc++;
        }
        return *this;
    }

    __host__ __device__ sptr& operator=(sptr&& x)
    {
        if (this->base != x.base)
        {
            delete this->base;
            this->base = x.base;
            x.base = nullptr;
        }
        return *this;
    }
};

template <typename el>
struct csptr : public sptr<el>
{ // Shared pointer for closures specifically.
    using sptr<el>::sptr;
    template <typename... Args>
    __host__ __device__ auto operator()(Args... args) -> decltype(this->base->operator()(args...))
    {
        return this->base->operator()(args...);
    }
};

template <typename el, default_int max_length>
struct static_array
{
    el ptr[max_length];
    __host__ __device__ el& operator[](default_int i) {
        assert("The index has to be in range." && 0 <= i && i < max_length);
        return this->ptr[i];
    }
};

template <typename el, default_int max_length>
struct static_array_list
{
    default_int length{ 0 };
    el ptr[max_length];

    __host__ __device__ el& operator[](default_int i) {
        assert("The index has to be in range." && 0 <= i && i < this->length);
        return this->ptr[i];
    }
    __host__ __device__ void push(el& x) {
        ptr[this->length++] = x;
        assert("The array after pushing should not be greater than max length." && this->length <= max_length);
    }
    __host__ __device__ void push(el&& x) {
        ptr[this->length++] = std::move(x);
        assert("The array after pushing should not be greater than max length." && this->length <= max_length);
    }
    __host__ __device__ el pop() {
        assert("The array before popping should be greater than 0." && 0 < this->length);
        auto x = ptr[--this->length];
        ptr[this->length].~el();
        new (&ptr[this->length]) el();
        return x;
    }
    // Should be used only during initialization.
    __host__ __device__ void unsafe_set_length(default_int i) {
        assert("The new length should be in range." && 0 <= i && i <= max_length);
        this->length = i;
    }
};

template <typename el, default_int max_length>
struct dynamic_array_base
{
    int refc{ 0 };
    el* ptr;

    __host__ __device__ dynamic_array_base() : ptr(new el[max_length]) {}
    __host__ __device__ ~dynamic_array_base() { delete[] this->ptr; }

    __host__ __device__ el& operator[](default_int i) {
        assert("The index has to be in range." && 0 <= i && i < this->length);
        return this->ptr[i];
    }
};

template <typename el, default_int max_length>
struct dynamic_array
{
    sptr<dynamic_array_base<el, max_length>> ptr;

    __host__ __device__ dynamic_array() = default;
    __host__ __device__ dynamic_array(bool t) : ptr(new dynamic_array_base<el, max_length>()) {}
    __host__ __device__ el& operator[](default_int i) {
        return this->ptr.base->operator[](i);
    }
};

template <typename el, default_int max_length>
struct dynamic_array_list_base
{
    int refc{ 0 };
    default_int length{ 0 };
    el* ptr;

    __host__ __device__ dynamic_array_list_base() : ptr(new el[max_length]) {}
    __host__ __device__ dynamic_array_list_base(default_int l) : ptr(new el[max_length]) { this->unsafe_set_length(l); }
    __host__ __device__ ~dynamic_array_list_base() { delete[] this->ptr; }

    __host__ __device__ el& operator[](default_int i) {
        assert("The index has to be in range." && 0 <= i && i < this->length);
        return this->ptr[i];
    }
    __host__ __device__ void push(el& x) {
        ptr[this->length++] = x;
        assert("The array after pushing should not be greater than max length." && this->length <= max_length);
    }
    __host__ __device__ void push(el&& x) {
        ptr[this->length++] = std::move(x);
        assert("The array after pushing should not be greater than max length." && this->length <= max_length);
    }
    __host__ __device__ el pop() {
        assert("The array before popping should be greater than 0." && 0 < this->length);
        auto x = ptr[--this->length];
        ptr[this->length].~el();
        new (&ptr[this->length]) el();
        return x;
    }
    // Should be used only during initialization.
    __host__ __device__ void unsafe_set_length(default_int i) {
        assert("The new length should be in range." && 0 <= i && i <= max_length);
        this->length = i;
    }
};

template <typename el, default_int max_length>
struct dynamic_array_list
{
    sptr<dynamic_array_list_base<el, max_length>> ptr;

    __host__ __device__ dynamic_array_list() = default;
    __host__ __device__ dynamic_array_list(default_int l) : ptr(new dynamic_array_list_base<el, max_length>(l)) {}

    __host__ __device__ el& operator[](default_int i) {
        return this->ptr.base->operator[](i);
    }
    __host__ __device__ void push(el& x) {
        this->ptr.base->push(x);
    }
    __host__ __device__ void push(el&& x) {
        this->ptr.base->push(std::move(x));
    }
    __host__ __device__ el pop() {
        return this->ptr.base->pop();
    }
    // Should be used only during initialization.
    __host__ __device__ void unsafe_set_length(default_int i) {
        this->ptr.base->unsafe_set_length(i);
    }
    __host__ __device__ default_int length_() {
        return this->ptr.base->length;
    }
};
