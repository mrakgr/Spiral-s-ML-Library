#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t error, const char *file, int line, bool abort=true) {
    if (error != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(error), file, line);
        if (abort) exit(error);
    }
}

__global__ void __cluster_dims__(8) hello(int a, int b) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello, CUDA! %i + %i = %i\n", a, b, a + b);
    }
}

int main() {
    size_t maxDynamicSharedMemory = 114 * (1 << 10);
    hipError_t error = hipFuncSetAttribute(reinterpret_cast<const void*>(hello), hipFuncAttributeMaxDynamicSharedMemorySize, maxDynamicSharedMemory);

    int i = 0;
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    void * args[] = {reinterpret_cast<void*>(&i), reinterpret_cast<void*>(&i)};
    gpuErrchk(hipLaunchCooperativeKernel(reinterpret_cast<const void*>(hello), 128, 256, args, maxDynamicSharedMemory));
    gpuErrchk(hipDeviceSynchronize());
    std::cout << "Done." << std::endl;
    return 0;
}

/*
pwsh tests/native_cuda/compile.ps1
*/