#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t error, const char *file, int line, bool abort=true) {
    if (error != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(error), file, line);
        if (abort) exit(error);
    }
}

__global__ void __cluster_dims__(16) hello(int a, int b) {
    if (threadIdx.x == 0) {
        printf("Hello, CUDA from %i! %i + %i = %i\n", blockIdx.x, a, b, a + b);
    }
}

int main() {
    size_t maxDynamicSharedMemory = 214 * (1 << 10);
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(hello), hipFuncAttributeMaxDynamicSharedMemorySize, maxDynamicSharedMemory));
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(hello), cudaFuncAttributeNonPortableClusterSizeAllowed, 16));

    int i = 0;
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    void * args[] = {reinterpret_cast<void*>(&i), reinterpret_cast<void*>(&i)};
    gpuErrchk(hipLaunchCooperativeKernel(reinterpret_cast<const void*>(hello), 112, 256, args, maxDynamicSharedMemory));
    gpuErrchk(hipDeviceSynchronize());
    std::cout << "Done." << std::endl;
    return 0;
}

/*
pwsh tests/native_cuda/compile.ps1
*/